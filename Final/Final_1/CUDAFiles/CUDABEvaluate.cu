#include "hip/hip_runtime.h"
#pragma once
#include "cutil_math.h"
#include "CUDAHelper.h"

__global__ void CUDABasisEvalKernel(float* knotData, int k, int kVal, int evalNum, float* basisData)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i >= (kVal - k) && i <= kVal && j < evalNum)
	{
		int index = j*kVal + i;
		float d = 0;
		float e = 0;
		int knotIndex = j*(2*kVal+1) + i;
		float u			= knotData[knotIndex - i];
		float prevI		= basisData[index];
		float prevI1	= 0.0;
		if (i < kVal-1)
			prevI1 = basisData[index + 1];
		
		float knotI		= knotData[knotIndex + 1];
		float knotIK1	= knotData[knotIndex + k];
		float knotIK = knotData[knotIndex + k + 1];
		float knotI1 = knotData[knotIndex + 2];
		if ( prevI!= 0 && knotIK1 != knotI)
			d = ((u-knotI)*prevI)/(knotIK1-knotI);
		if ( prevI1!= 0 && knotIK != knotI1)
			e = ((knotIK-u)*prevI1)/(knotIK-knotI1);
		basisData[index] = d + e;
	}
}

// Wrapper for the __global__ call that sets up the kernel call
extern "C" int CUDABasisEvaluate(float* knotData, int pass, int kVal, int evalNum, float* basisData)
{
    // execute the kernel
    dim3 block(GetBlockSize(kVal,4), GetBlockSize(evalNum,64), 1);
	dim3 grid(GetGridSize(kVal,block.x), GetGridSize(evalNum,block.y), 1);
	if ((block.x*block.y < 512) && (grid.x*grid.y < 65536))
		CUDABasisEvalKernel<<<grid, block>>>(knotData, pass, kVal, evalNum, basisData);
	else
		return 0;
	return 1;
}