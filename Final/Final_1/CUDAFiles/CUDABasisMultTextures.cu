#include "hip/hip_runtime.h"
#pragma once
#include "cutil_math.h"
#include "CUDAHelper.h"

texture<float4, 2, hipReadModeElementType> ctrlPointsGPUTexture;
texture<float, 2, hipReadModeElementType> uBasisGPUTexture;
texture<float, 2, hipReadModeElementType> vBasisGPUTexture;
texture<int, 1, hipReadModeElementType> uCtrlDataGPUTexture;
texture<int, 1, hipReadModeElementType> vCtrlDataGPUTexture;

__global__ void CUDABasisMultKernelTextures(float4* surfPt, int ku, int kv, int nu, int uNum, int vNum, int uPass, int vPass)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i < uNum && j < vNum)
	{
		float uBasis		= tex2D(uBasisGPUTexture, uPass, i);
		float vBasis		= tex2D(vBasisGPUTexture, vPass, j);
		int uCtrlIndexPos	= tex1D(uCtrlDataGPUTexture,i) + uPass;
		int vCtrlIndexPos	= tex1D(vCtrlDataGPUTexture,j) + vPass;
		float4 ctrlPt		= tex2D(ctrlPointsGPUTexture, uCtrlIndexPos, vCtrlIndexPos);

		float4 surfacePt = make_float4(0,0,0,0);
		
		surfacePt.w = ctrlPt.w*uBasis*vBasis;
		surfacePt.x = ctrlPt.x*surfacePt.w;
		surfacePt.y = ctrlPt.y*surfacePt.w;
		surfacePt.z = ctrlPt.z*surfacePt.w;
		
		if (uPass==0 && vPass==0)
			surfPt[j*uNum+i] = surfacePt;
		else
			surfPt[j*uNum+i] += surfacePt;
	}
//	surfPt[j*uNum+i] = 
//		make_float4(ctrlPtx,ctrlPty,ctrlPtz,ctrlPtw);
//		make_float4(uBasis,vBasis,uCtrlIndexPos,vCtrlIndexPos);
}

__global__ void CUDABasisMultKernelTexturesNoPP(float4* surfPt, int ku, int kv, int nu, int uNum, int vNum)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i < uNum && j < vNum)
	{
		for(int vPass=0; vPass<kv; vPass++)
		{
			for(int uPass=0; uPass<ku; uPass++)
			{
				float uBasis		= tex2D(uBasisGPUTexture, uPass, i);
				float vBasis		= tex2D(vBasisGPUTexture, vPass, j);
				int uCtrlIndexPos	= tex1D(uCtrlDataGPUTexture,i) + uPass;
				int vCtrlIndexPos	= tex1D(vCtrlDataGPUTexture,j) + vPass;
				float4 ctrlPt		= tex2D(ctrlPointsGPUTexture, uCtrlIndexPos, vCtrlIndexPos);

				float4 surfacePt = make_float4(0,0,0,0);
				
				surfacePt.w = ctrlPt.w*uBasis*vBasis;
				surfacePt.x = ctrlPt.x*surfacePt.w;
				surfacePt.y = ctrlPt.y*surfacePt.w;
				surfacePt.z = ctrlPt.z*surfacePt.w;
				
				if (uPass==0 && vPass==0)
					surfPt[j*uNum+i] = surfacePt;
				else
					surfPt[j*uNum+i] += surfacePt;
			}
		}
	}
}

// Wrapper for the __global__ call that sets up the kernel call
extern "C" int CUDABasisMultTextures(hipArray* ctrlPtsArray, hipArray* uCtrlData, hipArray* vCtrlData, hipArray* uBasisData, hipArray* vBasisData, float4* surfPt, int uNum, int vNum, int nu, int nv, int ku, int kv, int uPass, int vPass)
{
	// Allocate array and copy Texture data
    hipChannelFormatDesc float4ChannelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    hipChannelFormatDesc float1ChannelDesc	= hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipChannelFormatDesc int1ChannelDesc	= hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);

    // Set texture parameters
    ctrlPointsGPUTexture.addressMode[0] = hipAddressModeClamp;
    ctrlPointsGPUTexture.addressMode[1] = hipAddressModeClamp;
    ctrlPointsGPUTexture.filterMode = hipFilterModePoint;
    ctrlPointsGPUTexture.normalized = false;

    uBasisGPUTexture.addressMode[0] = hipAddressModeClamp;
    uBasisGPUTexture.addressMode[1] = hipAddressModeClamp;
    uBasisGPUTexture.filterMode = hipFilterModePoint;
    uBasisGPUTexture.normalized = false;

    vBasisGPUTexture.addressMode[0] = hipAddressModeClamp;
    vBasisGPUTexture.addressMode[1] = hipAddressModeClamp;
    vBasisGPUTexture.filterMode = hipFilterModePoint;
    vBasisGPUTexture.normalized = false;

    uCtrlDataGPUTexture.addressMode[0] = hipAddressModeClamp;
    uCtrlDataGPUTexture.filterMode = hipFilterModePoint;
    uCtrlDataGPUTexture.normalized = false;

	vCtrlDataGPUTexture.addressMode[0] = hipAddressModeClamp;
    vCtrlDataGPUTexture.filterMode = hipFilterModePoint;
    vCtrlDataGPUTexture.normalized = false;

	// Bind the array to the texture
   hipBindTextureToArray(ctrlPointsGPUTexture, ctrlPtsArray, float4ChannelDesc);
   hipBindTextureToArray(uBasisGPUTexture, uBasisData, float1ChannelDesc);
   hipBindTextureToArray(vBasisGPUTexture, vBasisData, float1ChannelDesc);
   hipBindTextureToArray(uCtrlDataGPUTexture, uCtrlData, int1ChannelDesc);
   hipBindTextureToArray(vCtrlDataGPUTexture, vCtrlData, int1ChannelDesc);

	// execute the kernel
    dim3 block(GetBlockSize(uNum,16), GetBlockSize(vNum,16), 1);
	dim3 grid(GetGridSize(uNum,block.x), GetGridSize(vNum,block.y), 1);
	if (uPass >= 0 && vPass >= 0)
	    CUDABasisMultKernelTextures<<<grid, block>>>(surfPt, ku, kv, nu, uNum, vNum, uPass, vPass);
	else
	    CUDABasisMultKernelTexturesNoPP<<<grid, block>>>(surfPt, ku, kv, nu, uNum, vNum);
	return 1;
}